#include <cutil.h>
#include <hipfft/hipfft.h>
#include <sys/time.h>

#define NX 2 * 512 * 8192
#define NY 4 * 1024
#define BATCH 2

int main() {

    // Initialise stuff
    hipEvent_t event_start, event_stop;
    struct timeval start, stop;
    struct timezone tzp;
    int i;

    // Initialise CUDA stuff
    hipSetDevice(0);
    hipEventCreate(&event_start); 
    hipEventCreate(&event_stop); 
    float timestamp;

    // ==================== 1D FFT ==============================

    hipfftHandle plan;
    hipfftComplex *d_data, *data;

    // Initialise data 
    data = (hipfftComplex *) malloc(NX * BATCH * sizeof(hipfftComplex));
    for(i = 0; i < NX * BATCH; i++) {
        data[i].x = 1.0f;
        data[i].y = 1.0f;
    }

   // Allocate and transfer data to GPU
  hipMalloc((void **) &d_data, sizeof(hipfftComplex) * NX * BATCH);
   hipEventRecord(event_start, 0);
   hipMemcpy(d_data, data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);
    hipEventRecord(event_stop, 0);
   hipEventSynchronize(event_stop);
   hipEventElapsedTime(&timestamp, event_start, event_stop);
   printf("Copied input to GPU in: %lfms\n", timestamp);

   // Create plan
   hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH);

   // Execute FFT
   hipEventRecord(event_start, 0);
   hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
   hipEventRecord(event_stop, 0);
   hipEventSynchronize(event_stop);
   hipEventElapsedTime(&timestamp, event_start, event_stop);
   printf("Processed 1D FFT in: %lfms\n", timestamp);
 
   // Get result
   hipMemcpy(data, d_data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost);

   // Clean up
   hipfftDestroy(plan);
   hipFree(d_data);

 // ==================== 2D FFT ==============================

//    // Initialise data 
//    realloc(data, NX * BATCH * sizeof(hipfftComplex));
//    for(i = 0; i < NX * BATCH; i++) {
//        data[i].x = 1.0f;
//        data[i].y = 1.0f;
//    }

//   // Allocate and transfer data to GPU
//   hipMalloc((void **) &d_data, sizeof(hipfftComplex) * NX * BATCH);
//   hipMemcpy(d_data, data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);

//   // Create plan
//   hipfftPlan2d(&plan, NY, NY, HIPFFT_C2C);

//   // Execute FFT
//   gettimeofday(&start, &tzp);
//   hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
//   hipDeviceSynchronize();
//   gettimeofday(&stop, &tzp);
//   if (start.tv_usec > stop.tv_usec) {
//       stop.tv_usec += 1000000;
//       stop.tv_sec--;
//   }
//   printf("Processed 2D FFT in %0.2fms [%d, %d]\n", (stop.tv_usec - start.tv_usec) / 1000.0f, NY, NY);
// 
//   // Get result
//   hipMemcpy(data, d_data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost);

   // Clean up
   hipfftDestroy(plan);
   hipFree(d_data);

}
